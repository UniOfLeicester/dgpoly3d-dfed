#include "hip/hip_runtime.h"
#include "calcs.h"
#include "types.h"
#include "kernel_assembleElems.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime_api.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


std::vector<Real> calcs(int NT,
                        int Nbasis,
                        int Ngauss3,
                        std::vector<Real> nodes,
                        std::vector<Real> mbb,
                        std::vector<int> tetrahedrons,
                        std::vector<int> tetrahedrons2elem,
                        std::vector<int> NbasisCummulative,
                        std::vector<Real> Aval,
                        std::vector<int> Aindices,
                        std::vector<int> Aindptr,
                        std::vector<Real> legendreCoefs,
                        std::vector<Real> nw_elem,
                        std::vector<int> basisCombinations)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    Real (*d_nodes)[3];
    hipMalloc(&d_nodes, nodes.size() * sizeof(decltype(nodes)::value_type));
    hipMemcpy(d_nodes, nodes.data(), nodes.size() * sizeof(decltype(nodes)::value_type), hipMemcpyHostToDevice);

    Real (*d_mbb)[6];
    hipMalloc(&d_mbb, mbb.size() * sizeof(decltype(mbb)::value_type));
    hipMemcpy(d_mbb, mbb.data(), mbb.size() * sizeof(decltype(mbb)::value_type), hipMemcpyHostToDevice);

    int (*d_tetrahedrons)[4];
    hipMalloc(&d_tetrahedrons, tetrahedrons.size() * sizeof(decltype(tetrahedrons)::value_type));
    hipMemcpy(d_tetrahedrons, tetrahedrons.data(), tetrahedrons.size() * sizeof(decltype(tetrahedrons)::value_type), hipMemcpyHostToDevice);

    int *d_tetrahedrons2elem;
    hipMalloc(&d_tetrahedrons2elem, tetrahedrons2elem.size() * sizeof(decltype(tetrahedrons2elem)::value_type));
    hipMemcpy(d_tetrahedrons2elem, tetrahedrons2elem.data(), tetrahedrons2elem.size() * sizeof(decltype(tetrahedrons2elem)::value_type), hipMemcpyHostToDevice);

    int *d_NbasisCummulative;
    hipMalloc(&d_NbasisCummulative, NbasisCummulative.size() * sizeof(decltype(NbasisCummulative)::value_type));
    hipMemcpy(d_NbasisCummulative, NbasisCummulative.data(), NbasisCummulative.size() * sizeof(decltype(NbasisCummulative)::value_type), hipMemcpyHostToDevice);

    Real *d_Aval;
    hipMalloc(&d_Aval, Aval.size() * sizeof(decltype(Aval)::value_type));
    hipMemcpy(d_Aval, Aval.data(), Aval.size() * sizeof(decltype(Aval)::value_type), hipMemcpyHostToDevice);

    int *d_Aindices;
    hipMalloc(&d_Aindices, Aindices.size() * sizeof(decltype(Aindices)::value_type));
    hipMemcpy(d_Aindices, Aindices.data(), Aindices.size() * sizeof(decltype(Aindices)::value_type), hipMemcpyHostToDevice);

    int *d_Aindptr;
    hipMalloc(&d_Aindptr, Aindptr.size() * sizeof(decltype(Aindptr)::value_type));
    hipMemcpy(d_Aindptr, Aindptr.data(), Aindptr.size() * sizeof(decltype(Aindptr)::value_type), hipMemcpyHostToDevice);


    // Set the constant memory
    elems_set_constant_mem(legendreCoefs, nw_elem, basisCombinations);

    // Launch the kernel
    int gridSize, blockSize;
    blockSize = 128;
    gridSize = ceil(NT / (float) blockSize);
    // printf("%d - %d\n", gridSize, blockSize);

    hipEventRecord(start);
    assembleElems<<<gridSize, blockSize>>>(NT, Nbasis, Ngauss3,
                                           d_nodes, d_mbb, d_tetrahedrons, d_tetrahedrons2elem,
                                           d_NbasisCummulative, d_Aval, d_Aindices, d_Aindptr);
    // gpuErrchk(hipPeekAtLastError());
    // gpuErrchk(hipDeviceSynchronize());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for kernel assembleElems (ms): %.2f\n", milliseconds);



    // Fetch bach the values of A
    std::vector<Real> Aval_elems(Aval.size());
    hipMemcpy(Aval_elems.data(), d_Aval, Aval.size() * sizeof(decltype(Aval)::value_type), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_nodes);
    hipFree(d_mbb);
    hipFree(d_tetrahedrons);
    hipFree(d_tetrahedrons2elem);
    hipFree(d_NbasisCummulative);
    hipFree(d_Aval);
    hipFree(d_Aindices);
    hipFree(d_Aindptr);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return Aval_elems;

}
